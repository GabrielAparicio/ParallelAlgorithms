
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define TILE 32
#define THREAD_PER_BLOCK TILE*TILE

__global__
void add_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;

	if ( col<n && row<n )
	{
		c[row*n+col] = a[row*n+col] + b[row*n+col];
	}
}

__global__
void mult_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;


	if ( col<n && row<n )
	{
		int i;
		c[row*n+col] = 0;

		for(i=0;i<n;i++)
		{
			c[row*n + col] += a[ row*n + i ]*b[ i*n + col ];

		}

	}
}

__global__
void mult_matrix_shared(int* a, int* b, int* c,int n)
{
	__shared__ float sub_a[TILE][TILE];
	__shared__ float sub_b[TILE][TILE];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int Row = by * TILE + ty;
	int Col = bx * TILE + tx;

	int Pvalue = 0;
	
	for (int ph = 0; ph < n/TILE; ++ph) {
	
		sub_a[ty][tx] = a[Row*n + ph*TILE + tx];
		sub_b[ty][tx] = b[(ph*TILE + ty)*n + Col];
		__syncthreads();
		
		for (int k = 0; k < TILE; ++k) {
			Pvalue += sub_a[ty][k] * sub_b[k][tx];
		}
		__syncthreads();
	}
	c[Row*n + Col] = Pvalue;
}

void print_matrix(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			printf("%d ",a[i*n+j]);
		}
		printf("\n");
	}
}

void fill_mat(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			a[i*n+j] = rand()%5+1;
		}
	}
}

int main()
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;

	int mat_elem = 800;
	int my_size = mat_elem*mat_elem*sizeof(int);

	hipEvent_t my_start,my_stop;
	hipEventCreate(&my_start);
	hipEventCreate(&my_stop);

	a = (int*) malloc(my_size);
	b = (int*) malloc(my_size);
	c = (int*) malloc(my_size);

	fill_mat(a,mat_elem);
	fill_mat(b,mat_elem);
	
	hipMalloc((void**)&d_a,my_size);
	hipMalloc((void**)&d_b,my_size);
	hipMalloc((void**)&d_c,my_size);

	hipMemcpy(d_a,a,my_size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,my_size,hipMemcpyHostToDevice);

	dim3 my_block(THREAD_PER_BLOCK,THREAD_PER_BLOCK);
	dim3 my_grid((mat_elem + THREAD_PER_BLOCK-1)/my_block.x,(mat_elem + THREAD_PER_BLOCK-1)/my_block.y);
	
	//////////////////////ELAPSED TIME ///////////////////////////////
	
	hipEventRecord(my_start,0);
	//mult_matrix<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
	mult_matrix_shared<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
	hipEventRecord(my_stop,0);
	hipEventSynchronize(my_stop);
	/////////////////////////////////////////////////////
	
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time,my_start,my_stop);

	hipMemcpy(c,d_c,my_size,hipMemcpyDeviceToHost);
	
	printf("Execution time: %f\n",elapsed_time);
	return 0;
}

