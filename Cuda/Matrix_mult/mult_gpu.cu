
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define THREAD_PER_BLOCK 50

__global__
void add_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;

	if ( col<n && row<n )
	{
		c[row*n+col] = a[row*n+col] + b[row*n+col];
	}
}

__global__
void mult_matrix(int* a, int* b, int* c,int n)
{
	int col = blockDim.x*blockIdx.x+ threadIdx.x;
	int row = blockDim.y*blockIdx.y+ threadIdx.y;


	if ( col<n && row<n )
	{
		int i;
		c[row*n+col] = 0;

		for(i=0;i<n;i++)
		{
			c[row*n + col] += a[ row*n + i ]*b[ i*n + col ];

		}

	}
}

__global__
void mult_matrix_shared(int* a, int* b, int* c,int n)
{
	__shared__ float sub_a[THREAD_PER_BLOCK][THREAD_PER_BLOCK];
	__shared__ float sub_b[THREAD_PER_BLOCK][THREAD_PER_BLOCK];

	int bx = blockIdx.x; int by = blockIdx.y;
	int tx = threadIdx.x; int ty = threadIdx.y;
	
	int Row = by * THREAD_PER_BLOCK + ty;
	int Col = bx * THREAD_PER_BLOCK + tx;

	int Pvalue = 0;
	
	for (int ph = 0; ph < n/THREAD_PER_BLOCK; ++ph) {
	
		sub_a[ty][tx] = a[Row*n + ph*THREAD_PER_BLOCK + tx];
		sub_b[ty][tx] = b[(ph*THREAD_PER_BLOCK + ty)*n + Col];
		__syncthreads();
		
		for (int k = 0; k < THREAD_PER_BLOCK; ++k) {
			Pvalue += sub_a[ty][k] * sub_b[k][tx];
		}
		__syncthreads();
	}
	c[Row*n + Col] = Pvalue;
}

void print_matrix(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			printf("%d ",a[i*n+j]);
		}
		printf("\n");
	}
}

void fill_mat(int* a,int n)
{
	int i,j;
	for(i=0;i<n;i++)
	{
		for(j=0;j<n;j++)
		{
			a[i*n+j] = rand()%5+1;
		}
	}
}

int main()
{
	int *a,*b,*c;
	int *d_a,*d_b,*d_c;

	int mat_elem = 800;
	int my_size = mat_elem*mat_elem*sizeof(int);

	hipEvent_t my_start,my_stop;
	hipEventCreate(&my_start);
	hipEventCreate(&my_stop);

	a = (int*) malloc(my_size);
	b = (int*) malloc(my_size);
	c = (int*) malloc(my_size);

	fill_mat(a,mat_elem);
	fill_mat(b,mat_elem);
	
	hipMalloc((void**)&d_a,my_size);
	hipMalloc((void**)&d_b,my_size);
	hipMalloc((void**)&d_c,my_size);

	hipMemcpy(d_a,a,my_size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,b,my_size,hipMemcpyHostToDevice);

	dim3 my_block(THREAD_PER_BLOCK,THREAD_PER_BLOCK);
	dim3 my_grid((mat_elem + THREAD_PER_BLOCK-1)/my_block.x,(mat_elem + THREAD_PER_BLOCK-1)/my_block.y);

	//////////////////////ELAPSED TIME ///////////////////////////////
  hipEventRecord(my_start,0);
  //mult_matrix<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
  mult_matrix_shared<<<my_grid,my_block>>>(d_a, d_b, d_c,mat_elem);
  hipEventRecord(my_stop,0);
  hipEventSynchronize(my_stop);
  /////////////////////////////////////////////////////
  
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time,my_start,my_stop);

	hipMemcpy(c,d_c,my_size,hipMemcpyDeviceToHost);
	
	printf("Execution time: %f\n",elapsed_time);
	return 0;
}

