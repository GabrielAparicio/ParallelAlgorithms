
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define m 10
#define n 5

//  matrix_sum1<<<m, n>>>(d_A, d_B, d_C, m, n);
__global__ void matrix_sum1(int A[], int B[], int C[], int fil, int col) 
{
   
   int index = blockDim.x * blockIdx.x + threadIdx.x;

   if (blockIdx.x < fil && threadIdx.x < col) 
      C[index] = A[index] + B[index];
} 

// matrix_sum2<<<m, 1>>>(d_A, d_B, d_C, m, n);
__global__ void matrix_sum2(int A[], int B[], int C[], int fil, int col) 
{
   int index = blockIdx.x;
   int i;
   
   if(index < fil)
   {
   for(i=0;i<col;i++)
   { 
         C[index + i] = A[index + i] + B[index + i];
   }
   }
}


// matrix_sum2<<<n, 1>>>(d_A, d_B, d_C, m, n);
__global__ void matrix_sum3(int A[], int B[], int C[], int fil, int col) 
{
   int index = blockIdx.x;
   int i;
   
   if(index<col)
   {
      for(i=0;i<fil;i++)
      {
         C[col*i + i] = A[col*i+i] + B[col*i+i];
      }
   }
   
   
} 


void fill_matrix(int A[], int fil, int col) {
   int i, j;

   for (i = 0; i < fil; i++) {
      for (j = 0; j < col; j++)
         A[i*n+j] = rand()%99;
   } 
}

void print_matrix(int A[], int fil, int col) {
   int i, j;

   for (i = 0; i < fil; i++) {
      for (j = 0; j < col; j++)
         printf("%d ", A[i*n+j]);
      printf("\n");
   }  
}

int main(int argc, char* argv[]) {

   int *h_A, *h_B, *h_C;
   int *d_A, *d_B, *d_C;
   size_t size;

   
   size = m*n*sizeof(int);

   h_A = (int*) malloc(size);
   h_B = (int*) malloc(size);
   h_C = (int*) malloc(size);
   
   fill_matrix(h_A, m, n);
   fill_matrix(h_B, m, n);

   print_matrix(h_A, m, n);
   printf("\n");
   print_matrix(h_B, m, n);
   printf("\n");
   
   hipMalloc((void **)&d_A, size);
   hipMalloc((void **)&d_B, size);
   hipMalloc((void **)&d_C, size);

   
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   matrix_sum1<<<m, n>>>(d_A, d_B, d_C, m, n);

   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   print_matrix(h_C, m, n);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
} 
