
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define m 10
#define n 5


__global__ void matrix_sum(int A[], int B[], int C[], int fil, int col) 
{
   
   int my_ij = blockDim.x * blockIdx.x + threadIdx.x;

   if (blockIdx.x < fil && threadIdx.x < col) 
      C[my_ij] = A[my_ij] + B[my_ij];
} 



void fill_matrix(int A[], int fil, int col) {
   int i, j;

   for (i = 0; i < fil; i++) {
      for (j = 0; j < col; j++)
         A[i*n+j] = rand()%99;
   } 
}

void print_matrix(int A[], int fil, int col) {
   int i, j;

   for (i = 0; i < fil; i++) {
      for (j = 0; j < col; j++)
         printf("%d ", A[i*n+j]);
      printf("\n");
   }  
}

int main(int argc, char* argv[]) {

   int *h_A, *h_B, *h_C;
   int *d_A, *d_B, *d_C;
   size_t size;

   
   size = m*n*sizeof(int);

   h_A = (int*) malloc(size);
   h_B = (int*) malloc(size);
   h_C = (int*) malloc(size);
   
   fill_matrix(h_A, m, n);
   fill_matrix(h_B, m, n);

   print_matrix(h_A, m, n);
   printf("\n");
   print_matrix(h_B, m, n);
   printf("\n");
   
   hipMalloc((void **)&d_A, size);
   hipMalloc((void **)&d_B, size);
   hipMalloc((void **)&d_C, size);

   
   hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
   hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

   matrix_sum<<<m, n>>>(d_A, d_B, d_C, m, n);

   hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

   print_matrix(h_C, m, n);

   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

   free(h_A);
   free(h_B);
   free(h_C);

   return 0;
} 
